#include "hip/hip_runtime.h"
#include "mex.h"
#include "gpu/mxGPUArray.h"

using namespace std;
void __global__ calculateImRe(double* idata,double* qdata,double* Im1,double* Re1,double* Im2,double* Re2,
        int depth,int width,int na)
{
    int id = (blockIdx.x * blockDim.x + threadIdx.x);
	if (id < depth * width * (na - 1))
	{   
		Im1[id] = qdata[id] * idata[id + depth * width] - \
			idata[id] * qdata[id + depth * width];
		Re1[id] = idata[id] * idata[id + depth * width] + \
			qdata[id] * qdata[id + depth * width];
	}
	if (id < (depth - 1) * width * na)
	{
        int i = id % (depth - 1);
		int j = id / (depth - 1);
        id= j * depth + i;
		Im2[id] = qdata[id] * idata[id + 1] - \
			idata[id] * qdata[id + 1];
		Re2[id] = idata[id] * idata[id + 1] + \
			qdata[id] * qdata[id + 1];
	}
}

__global__ void caculate_dis_withM(double* Im1, double* Re1, double* Im2, double* Re2,
	double* Im1_M, double* Re1_M, double* Im2_M, double* Re2_M, int depth, int width, int na, int M,  int N)
{
	int id = (blockIdx.x * blockDim.x + threadIdx.x);
	if (0 <= id && id < (depth - M + 1) * width * (na - 1))
	{
		int i = id % (depth - M + 1);
		int j = id / (depth - M + 1);
        double Im1_sum = 0.0;
		double Re1_sum = 0.0;
		for (int m = 0; m < M; ++m)
		{
            Im1_sum += Im1[j * depth + i + m];
            Re1_sum += Re1[j * depth + i + m];
		}
        Im1_M[j * depth + i] = Im1_sum;
        Re1_M[j * depth + i] = Re1_sum;
    }
    
    id = id - (depth - M + 1) * width * (na - 1);
    if (0 <= id && id < (depth - M + 1) * width * na)
	{
        int i = id % (depth - M + 1);
		int j = id / (depth - M + 1);
        double Im2_sum = 0.0;
		double Re2_sum = 0.0;
		for (int m = 0; m < M - 1; ++m)
		{
            Im2_sum += Im2[j * depth + i + m];
            Re2_sum += Re2[j * depth + i + m];
		}
        Im2_M[j * depth + i] = Im2_sum;
        Re2_M[j * depth + i] = Re2_sum;
	}
}
__global__ void caculate_dis_withN(double* Im1_N, double* Re1_N, double* Im2_N, double* Re2_N,
	double* Im1_M, double* Re1_M, double* Im2_M, double* Re2_M, int depth, int width, int na, int M,  int N)
{
	int id = (blockIdx.x * blockDim.x + threadIdx.x);
    
	if (0 <= id && id < (depth - M + 1) * width * (na - N + 1 ))
	{
		int i = id % (depth - M + 1);
		int j = id / (depth - M + 1) % width;
        int k = id / (depth - M + 1) / width;
		id = k * depth * width + j * depth + i;
        double Im1_sum = 0.0;
		double Re1_sum = 0.0;
		for (int n = 0; n < N - 1; ++n)
		{
            Im1_sum += Im1_M[id + n * depth * width];
            Re1_sum += Re1_M[id + n * depth * width];
		}   
        Im1_N[id] = Im1_sum;
        Re1_N[id] = Re1_sum;
    }
    id = id - (depth - M + 1) * width * (na - N +1);
    if (0 <= id && id < (depth - M + 1) * width * (na - N + 1))
	{
        int i = id % (depth - M + 1);
		int j = id / (depth - M + 1) % width;
        int k = id / (depth - M + 1) / width;
        id = k * depth * width + j * depth + i;
        double Im2_sum = 0.0;
		double Re2_sum = 0.0;
		for (int n = 0; n < N; ++n)
		{
            Im2_sum += Im2_M[id + n * depth * width];
            Re2_sum += Re2_M[id + n * depth * width];
		}
        Im2_N[id] = Im2_sum;
        Re2_N[id] = Re2_sum;
    }
}
__global__ void caculate_dis_without_conv(double* Dis, double* Im1, double* Re1, double* Im2, double* Re2,
	int depth, int width, int na, int M,  int N, double c, double fc)
{
	int id = (blockIdx.x * blockDim.x + threadIdx.x);
	if (id < (depth - M + 1) * width * (na - N + 1 ))
	{
		int i = id % (depth - M + 1);
		int j = id / (depth - M + 1) % width;
        int k = id / (depth - M + 1) / width;
        int id1 = k * depth * width + j * depth + i;
		Dis[id] = atan2(Im1[id1], Re1[id1]) / (1 + atan2(Im2[id1], Re2[id1]) / (2.0 * 3.1416)) * c / (4.0 * 3.1416 * fc);
    }
}
__global__ void caculate_dis_with_conv(double* Dis_k2D, double* Dis,
	int depth, int width, int na)
{
	double kernel2D[5][5] = {
		{ 0.0073, 0.0208, 0.0294, 0.0208, 0.0073 },
		{ 0.0208, 0.0589, 0.0833, 0.0589, 0.0208 },
		{ 0.0294, 0.0833, 0.1179, 0.0833, 0.0294 },
		{ 0.0208, 0.0589, 0.0833, 0.0589, 0.0208 },
		{ 0.0073, 0.0208, 0.0294, 0.0208, 0.0073 }
	};
	int id = (blockIdx.x * blockDim.x + threadIdx.x);
	if (id < depth * width * na)
	{
		int i = id % depth;
		int j = id / depth % width;
		double sum = 0.0;
		for (int m = -2; m < 3; ++m)
		{
			for (int n = -2; n < 3; ++n)
			{
				if (i + m >= 0 && i + m < depth && j + n >= 0 && j + n < width)
				{
					sum += kernel2D[m + 2][n + 2] * Dis[id + n * depth + m];
				}
			}
		}
		Dis_k2D[id] = sum;
	}
}
__global__ void caculate_ave_dis_in_axial(double* Dis_aver, double* Dis_k2D,
    int averN, int depth, int width, int na)
{
	int id = (blockIdx.x * blockDim.x + threadIdx.x);
	if (id < depth * width * na)
	{
		int i = id % depth;
		double sum = 0.0;
		for (int m = (-averN + 1) / 2; m <= averN / 2; ++m)
		{
			if (i + m >= 0 && i + m < depth)
			{
				sum += Dis_k2D[id + m];
			}
		}
		Dis_aver[id] = sum / averN;
	}
}

__global__ void caculate_min_index(double* minIndex, double* Dis,
	int depth, int width, int na)
{
	int id = (blockIdx.x * blockDim.x + threadIdx.x);
	if (id < depth * width)
	{
		double minValue = 0;
		double minInd = 0;
		for (int k = 0; k < na; ++k)
		{
			if (Dis[id + k * depth * width] < minValue)
			{
				minValue = Dis[id + k * depth * width];
				minInd = (double)k;
			}
		}
		minIndex[id] = minInd + 1;
	}
}

// __device__ double polyfit_wt(double* points, int pointNum)
// {   
//     double t1=0.0;
//     double t2=0.0;
//     double t3=0.0;
//     double t4=0.0;
//     for (int i = 0; i < pointNum; ++i)
//     {
//         t1 += points[2 * i] * points[2 * i];
//         t2 += points[2 * i];
//         t3 += points[2 * i + 1] * points[2 * i];
//         t4 += points[2 * i + 1];
//     }
//     return (t3 * pointNum - t2 * t4) / (pointNum * t1 - t2 * t2);
// }

// __device__ double fitFunck(double x1, double y1, double x2, double y2)
// {
//     return (y2 - y1) / (x2 - x1);
// }
// 
// __device__ double fitFuncb(double x1, double y1, double x2, double y2)
// {
//     return (x2 * y1 - x1 * y2) / (x2 - x1);
// }

// __device__ int computeLoopNumber(double confidence, int pointNum, int inlierNum)
// {
//     double inlierProbability = (inlierNum / pointNum) * (inlierNum / pointNum);
//     double conf = 0.01 * confidence;
//     double num = log10(1 - conf);
//     double den = log10(1 - inlierProbability);
//     return (int)ceil(num / den);
// }

__device__ double msac_wt(double* points, int pointNum, double* randpoint, double maxDistance)
{   
    int numTrials  = 50;
    double bestDis = maxDistance * pointNum;
    double bestk = 0.0;
    double bestb = 0.0;
    double accDis = 0.0;
    double x1 = 0.0;
    double x2 = 0.0;
    double y1 = 0.0;
    double y2 = 0.0;
    double k  = 0.0;
    double b  = 0.0;
    double loss = 0.0;
    for (int j = 0; j < numTrials; ++j)
    {
        x1 = randpoint[j * 2];
        x2 = randpoint[j * 2 + 1];
        y1 = points[(int)x1];
        y2 = points[(int)x2];
        k  = (y2 - y1) / (x2 - x1);
        b  = (x2 * y1 - x1 * y2) / (x2 - x1);
        accDis = 0.0;
        for(int i = 0; i < pointNum; ++i)
        {
            loss=fabs((points[i]-k*i-b)/sqrt(k*k+b*b));
            accDis+=min(loss,maxDistance);
        }
        if (accDis < bestDis)
        {
            bestDis = accDis;
            bestk = k;
            bestb = b;
        }
    }
    double t1=0.0;
    double t2=0.0;
    double t3=0.0;
    double t4=0.0;
    int inlierNum = 0;
    for (int i = 0; i < pointNum; ++i)
    {
        loss=fabs((points[i]-bestk*i-bestb)/sqrt(bestk*bestk+bestb*bestb));
        if (loss < maxDistance)
        {
            inlierNum += 1;
            t1 += i*i;
            t2 += i;
            t3 += points[i] * i;
            t4 += points[i];
        }
    }
    return (t3 * inlierNum - t2 * t4) / (inlierNum * t1 - t2 * t2);
}


__global__ void caculate_Yang(double* Yang, double* minInd, double* randpoint, int pointNum,
	int Ledge, int Redge, int depth, double unitDistance, double unitTime)
{
    int id = (blockIdx.x * blockDim.x + threadIdx.x);
    if (id >= depth * Ledge && id < depth * Redge)
    {
        double* points = new double[pointNum];
        for (int i = 0; i < pointNum; ++i)
        {
            points[i]=minInd[id+(i-(pointNum-1)/2)*depth];
        }
        Yang[id]=pow(unitDistance/(unitTime * msac_wt(points,pointNum,randpoint,0.5)),2) * 3000;
        delete []points;
    }
}

void mexFunction(int nlhs, mxArray *plhs[],int nrhs,const mxArray *prhs[])
{   
    
    mxInitGPU();
    int const threadsPerBlock = 1024;
    mxGPUArray const *idata      = mxGPUCreateFromMxArray(prhs[0]);
    mxGPUArray const *qdata      = mxGPUCreateFromMxArray(prhs[1]);
    mxGPUArray const *randpoint  = mxGPUCreateFromMxArray(prhs[2]);
    double *parameter            = mxGetPr(prhs[3]);
    
    double* d_idata    = (double*)(mxGPUGetDataReadOnly(idata));
    double* d_qdata    = (double*)(mxGPUGetDataReadOnly(qdata));
    double* d_randpoint= (double*)(mxGPUGetDataReadOnly(randpoint));
    mxClassID      cid = mxGPUGetClassID(idata);
    mxComplexity   ccx = mxGPUGetComplexity(idata);
    const mwSize* dims = mxGPUGetDimensions(idata);
    int          depth = (int)dims[0];
    int          width = (int)dims[1];
    int             na = (int)dims[2];
    int              M = (int)parameter[0];
    int              N = (int)parameter[1];
    double           c = (double)parameter[2];
    double          fc = (double)parameter[3];
    int          averN = (int)parameter[4];
    int       pointNum = (int)parameter[5];
    int          Ledge = (int)parameter[6];
    int          Redge = (int)parameter[7];
    double unitDistance = (double)parameter[8];
    double     unitTime = (double)parameter[9];
    mwSize* dim1s=new mwSize[3];
    dim1s[0]=dims[0]-M+1;dim1s[1]=dims[1];dim1s[2]=dims[2]-N+1;
    mwSize* dim2s=new mwSize[2];
    dim2s[0]=dims[0]-M+1;dim2s[1]=dims[1];
    mxGPUArray* Im1 = mxGPUCreateGPUArray(3,dims,cid,ccx,MX_GPU_DO_NOT_INITIALIZE);
    mxGPUArray* Re1 = mxGPUCreateGPUArray(3,dims,cid,ccx,MX_GPU_DO_NOT_INITIALIZE);
    mxGPUArray* Im2 = mxGPUCreateGPUArray(3,dims,cid,ccx,MX_GPU_DO_NOT_INITIALIZE);
    mxGPUArray* Re2 = mxGPUCreateGPUArray(3,dims,cid,ccx,MX_GPU_DO_NOT_INITIALIZE);
    mxGPUArray* Im1_M = mxGPUCreateGPUArray(3,dims,cid,ccx,MX_GPU_DO_NOT_INITIALIZE);
    mxGPUArray* Re1_M = mxGPUCreateGPUArray(3,dims,cid,ccx,MX_GPU_DO_NOT_INITIALIZE);
    mxGPUArray* Im2_M = mxGPUCreateGPUArray(3,dims,cid,ccx,MX_GPU_DO_NOT_INITIALIZE);
    mxGPUArray* Re2_M = mxGPUCreateGPUArray(3,dims,cid,ccx,MX_GPU_DO_NOT_INITIALIZE);
    mxGPUArray* Im1_N = mxGPUCreateGPUArray(3,dims,cid,ccx,MX_GPU_DO_NOT_INITIALIZE);
    mxGPUArray* Re1_N = mxGPUCreateGPUArray(3,dims,cid,ccx,MX_GPU_DO_NOT_INITIALIZE);
    mxGPUArray* Im2_N = mxGPUCreateGPUArray(3,dims,cid,ccx,MX_GPU_DO_NOT_INITIALIZE);
    mxGPUArray* Re2_N = mxGPUCreateGPUArray(3,dims,cid,ccx,MX_GPU_DO_NOT_INITIALIZE);
    mxGPUArray* Dis = mxGPUCreateGPUArray(3,dim1s,cid,ccx,MX_GPU_DO_NOT_INITIALIZE);
    mxGPUArray* Dis_k2D = mxGPUCreateGPUArray(3,dim1s,cid,ccx,MX_GPU_DO_NOT_INITIALIZE);
    mxGPUArray* Dis_aver = mxGPUCreateGPUArray(3,dim1s,cid,ccx,MX_GPU_DO_NOT_INITIALIZE);
    mxGPUArray* minInd = mxGPUCreateGPUArray(2,dim2s,cid,ccx,MX_GPU_DO_NOT_INITIALIZE);
    mxGPUArray* Yang = mxGPUCreateGPUArray(2,dim2s,cid,ccx,MX_GPU_DO_NOT_INITIALIZE);
    double* d_Im1 = (double *)(mxGPUGetData(Im1));
    double* d_Re1 = (double *)(mxGPUGetData(Re1));
    double* d_Im2 = (double *)(mxGPUGetData(Im2));
    double* d_Re2 = (double *)(mxGPUGetData(Re2));
    double* d_Im1_M = (double *)(mxGPUGetData(Im1_M));
    double* d_Re1_M = (double *)(mxGPUGetData(Re1_M));
    double* d_Im2_M = (double *)(mxGPUGetData(Im2_M));
    double* d_Re2_M = (double *)(mxGPUGetData(Re2_M));
    double* d_Im1_N = (double *)(mxGPUGetData(Im1_N));
    double* d_Re1_N = (double *)(mxGPUGetData(Re1_N));
    double* d_Im2_N = (double *)(mxGPUGetData(Im2_N));
    double* d_Re2_N = (double *)(mxGPUGetData(Re2_N));
    double* d_Dis = (double *)(mxGPUGetData(Dis));
    double* d_Dis_k2D = (double *)(mxGPUGetData(Dis_k2D));
    double* d_Dis_aver = (double *)(mxGPUGetData(Dis_aver));
    double* d_minInd = (double *)(mxGPUGetData(minInd));
    double* d_Yang   = (double *)(mxGPUGetData(Yang));
    int blocksPerGrid = (mxGPUGetNumberOfElements (idata)-1) / threadsPerBlock + 1;
    calculateImRe<<<blocksPerGrid, threadsPerBlock>>>(d_idata,d_qdata,d_Im1,d_Re1,d_Im2,d_Re2,depth,width,na);
    caculate_dis_withM <<<2*blocksPerGrid, threadsPerBlock >>>(d_Im1, d_Re1, d_Im2, d_Re2, \
            d_Im1_M, d_Re1_M, d_Im2_M, d_Re2_M, depth, width, na, M, N);
    caculate_dis_withN <<<2*blocksPerGrid, threadsPerBlock >>>(d_Im1_N, d_Re1_N, d_Im2_N, d_Re2_N, \
            d_Im1_M, d_Re1_M, d_Im2_M, d_Re2_M, depth, width, na, M, N);
    caculate_dis_without_conv <<<blocksPerGrid, threadsPerBlock >>> (d_Dis, d_Im1_N, d_Re1_N, d_Im2_N, d_Re2_N, \
            depth, width, na, M, N, c, fc);
	caculate_dis_with_conv <<<blocksPerGrid, threadsPerBlock >>> (d_Dis_k2D, d_Dis, depth - M + 1, width, na - N + 1);
    caculate_ave_dis_in_axial <<<blocksPerGrid, threadsPerBlock >>> (d_Dis_aver, d_Dis_k2D, averN, depth - M + 1, width, na - N + 1);
    caculate_min_index <<<blocksPerGrid, threadsPerBlock >>> (d_minInd, d_Dis_aver, depth - M + 1, width, na - N + 1);
    caculate_Yang <<<blocksPerGrid, threadsPerBlock >>> (d_Yang, d_minInd, d_randpoint, pointNum, Ledge, Redge, depth - M + 1, unitDistance, unitTime);
    plhs[0] = mxGPUCreateMxArrayOnGPU(Yang);
    mxGPUDestroyGPUArray(idata);
    mxGPUDestroyGPUArray(qdata);
    mxGPUDestroyGPUArray(randpoint);
    mxGPUDestroyGPUArray(Im1);
    mxGPUDestroyGPUArray(Re1);
    mxGPUDestroyGPUArray(Im2);
    mxGPUDestroyGPUArray(Re2);
    mxGPUDestroyGPUArray(Im1_M);
    mxGPUDestroyGPUArray(Re1_M);
    mxGPUDestroyGPUArray(Im2_M);
    mxGPUDestroyGPUArray(Re2_M);
    mxGPUDestroyGPUArray(Im1_N);
    mxGPUDestroyGPUArray(Re1_N);
    mxGPUDestroyGPUArray(Im2_N);
    mxGPUDestroyGPUArray(Re2_N);
    mxGPUDestroyGPUArray(Dis);
    mxGPUDestroyGPUArray(Dis_k2D);
    mxGPUDestroyGPUArray(Dis_aver);
    delete []dim1s;
    delete []dim2s;
}